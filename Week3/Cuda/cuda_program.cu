    #include <hip/hip_runtime.h>
    #include <iostream>

    __global__ void add(int *a, int *b, int *c, int N) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < N) {
            c[idx] = a[idx] + b[idx];
        }
    }

    int main() {
        int N = 1000;
        int size = N * sizeof(int);
        int *a, *b, *c;
        int *d_a, *d_b, *d_c;

        // Allocate host memory
        a = (int*)malloc(size);
        b = (int*)malloc(size);
        c = (int*)malloc(size);

        // Initialize host arrays
        for (int i = 0; i < N; i++) {
            a[i] = i;
            b[i] = i;
        }

        // Allocate device memory
        hipMalloc((void**)&d_a, size);
        hipMalloc((void**)&d_b, size);
        hipMalloc((void**)&d_c, size);

        // Copy data from host to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        // Launch kernel
        add<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

        // Copy result from device to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

        // Print result
        for (int i = 0; i < 10; i++) {
            std::cout << c[i] << " ";
        }
        std::cout << std::endl;

        // Free memory
        free(a);
        free(b);
        free(c);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        return 0;
    }
